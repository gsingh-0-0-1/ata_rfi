#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <iostream>

typedef struct comp_float {
    float real;
    float imag;
} comp_float_t;


/*
// CUDA kernel to compute the mean
__global__ void computeChannelMean(
    const float* block, float* output,
    int N_ANTS, int N_CHANS, int N_SAMPS, int N_POLS) {

    // Compute indices
    int ant = blockIdx.x;    // Antenna index
    int chan = blockIdx.y;   // Channel index
    int pol = threadIdx.x;   // Polarization index

    if (ant < N_ANTS && chan < N_CHANS && pol < N_POLS) {
        // Compute the mean for this channel
        float sum = 0.0f;
        for (int samp = 0; samp < N_SAMPS; ++samp) {
            int idx = ((ant * N_CHANS + chan) * N_SAMPS + samp) * N_POLS + pol;
            sum += block[idx];
        }
        float mean = sum / N_SAMPS;

        // Write the result to the output array
        int out_idx = ((ant * N_CHANS + chan) * 1 + 0) * N_POLS + pol;
        output[out_idx] = mean;
    }
}

// Host function to call the kernel
void calculateMean(
    const float* d_block, float* d_output,
    int N_ANTS, int N_CHANS, int N_SAMPS, int N_POLS) {

    dim3 gridDim(N_ANTS, N_CHANS);    // One block per antenna and channel
    dim3 blockDim(N_POLS);           // One thread per polarization

    computeChannelMean<<<gridDim, blockDim>>>(
        d_block, d_output, N_ANTS, N_CHANS, N_SAMPS, N_POLS);
}
*/

// CUDA kernel to compute sk_array
__global__ void computeSkArray(
    comp_float_t* block,
    int N_ANTS, int N_CHANS, int N_SAMPS, int N_POLS) {//, int m) {


    // Compute indices
    int ant = blockIdx.x;    // Antenna index
    int chan = blockIdx.y;   // Channel index
    int pol = threadIdx.x;   // Polarization index

    if (ant < N_ANTS && chan < N_CHANS && pol < N_POLS) {
        // Initialize sums
        float s1 = 0.0f;
        float s2 = 0.0f;

        // Compute s1 (sum of elements) and s2 (sum of squares)
        for (int samp = 0; samp < N_SAMPS; samp++) {
            int idx = ((ant * N_CHANS + chan) * N_SAMPS + samp) * N_POLS + pol;
            comp_float_t value = block[idx];

            float v2 = value.real * value.real + value.imag * value.imag;

            s1 += v2;
            s2 += v2 * v2;
        }

        // Compute sk value
        float sk = ((N_SAMPS + 1.0f) / (N_SAMPS - 1.0f)) * ((N_SAMPS * (s2 / (s1 * s1))) - 1.0f);

        // based on sk we can zap the channel
        // TODO: change sk thresholds / properly apply from sklim
        if (sk > 3) {
            int chan_start = ((ant * N_CHANS + chan) * N_SAMPS + 0) * N_POLS + pol;
            for (int j = chan_start; j < chan_start + N_SAMPS * N_POLS; j = j + N_POLS) {
                block[j].real = 0.0;
                block[j].imag = 0.0;
            }
        }

        // Write the result to the output array
        // int out_idx = ((ant * N_CHANS + chan) * 1 + 0) * N_POLS + pol;
        // output[out_idx] = sk;
    }
}

// Host function to call the kernel
void calculateSkArray(
    comp_float_t* d_block,
    int N_ANTS, int N_CHANS, int N_SAMPS, int N_POLS) {//, int m) {

    dim3 gridDim(N_ANTS, N_CHANS);    // One block per antenna and channel
    dim3 blockDim(N_POLS);           // One thread per polarization

    computeSkArray<<<gridDim, blockDim>>>(
        d_block, N_ANTS, N_CHANS, N_SAMPS, N_POLS);//, m);
}


int main() {
    // Define dimensions
    int N_ANTS = 4, N_CHANS = 4, N_SAMPS = 8, N_POLS = 2;

    // Allocate memory on host
    size_t input_size = N_ANTS * N_CHANS * N_SAMPS * N_POLS * sizeof(comp_float_t);
    // size_t output_size = N_ANTS * N_CHANS * 1 * N_POLS * sizeof(float);
    comp_float_t* h_block = (comp_float_t*)malloc(input_size);
    // float* h_output = (float*)malloc(output_size);

    std::random_device rd{};
    std::mt19937 gen{rd()};

    std::normal_distribution d{0.0, 1.0};

    auto randnorm = [&d, &gen]{ return d(gen); };

    // Initialize input data
    for (int i = 0; i < N_ANTS * N_CHANS * N_SAMPS * N_POLS; ++i) {
        h_block[i].imag = static_cast<float>(randnorm());
        h_block[i].real = static_cast<float>(randnorm());
    }

    // pollute the first two channels
    for (int i = 0; i < 4; i++) {
        h_block[i].real = 100.0f;
    }


    // print block before
    for (int ant = 0; ant < N_ANTS; ant++) {
        for (int chan = 0; chan < N_CHANS; chan++) {
            for (int pol = 0; pol < N_POLS; pol++) {
                int chan_start = ((ant * N_CHANS + chan) * N_SAMPS + 0) * N_POLS + pol;
                for (int j = chan_start; j < chan_start + N_SAMPS * N_POLS; j = j + N_POLS) {
                    std::cout << h_block[j].real << " + " << h_block[j].imag << "i, ";
                }
                std::cout << std::endl;
            }
        }
    }

    std::cout << std::endl << std::endl;

    // Allocate memory on device
    comp_float_t *d_block;
    //float *d_output;
    hipMalloc(&d_block, input_size);
    //cudaMalloc(&d_output, output_size);

    // Copy data to device
    hipMemcpy(d_block, h_block, input_size, hipMemcpyHostToDevice);

    // Launch kernel
    calculateSkArray(d_block,  N_ANTS, N_CHANS, N_SAMPS, N_POLS);

    // Copy result back to host
    hipMemcpy(h_block, d_block, input_size, hipMemcpyDeviceToHost);
    //cudaMemcpy(h_output, d_output, output_size, cudaMemcpyDeviceToHost);

    // print block after
    for (int ant = 0; ant < N_ANTS; ant++) {
        for (int chan = 0; chan < N_CHANS; chan++) {
            for (int pol = 0; pol < N_POLS; pol++) {
                int chan_start = ((ant * N_CHANS + chan) * N_SAMPS + 0) * N_POLS + pol;
                for (int j = chan_start; j < chan_start + N_SAMPS * N_POLS; j = j + N_POLS) {
                    std::cout << h_block[j].real << " + " << h_block[j].imag << "i, ";
                }
                std::cout << std::endl;
            }
        }
    }

    //std::cout << d_block[0] << std::endl;

    //for (int i = 0; i < 20; i++) {
        //std::cout << h_block[i * 8] << " " << h_block[i * 8 + 2] << " " << h_block[i * 8 + 4] << " " << h_block[i * 8 + 6] << " " << h_output[i] << std::endl;
        //std::cout << h_output[i] << std::endl;
    //}

    // Cleanup
    hipFree(d_block);
    free(h_block);

    return 0;
}

